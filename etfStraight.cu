
#include <hip/hip_runtime.h>
#include <math.h>

const int LEN = 4;

// get a line centered at *ctrp, along the direction of *vec
__device__ void line2D(double out[][2*LEN+1], double *ctrp, double *vec) {
    double x = ctrp[1];
    double y = ctrp[0];
    double gx = vec[0];
    double gy = vec[1];
    for (int i = -LEN; i <= LEN; i++){
        out[0][i+LEN] = x+i*gx;
        out[1][i+LEN] = y+i*gy;
    }
}

// single point calculation
__device__ double dire_weight(double *x, double *y) {
    return y[0]*x[0]+y[1]*x[1];
}

__device__ double mag_weight(double x, double y) {
    return 0.5*(1+y-x);
}

__global__ void etfStraight(
                double *xout, double *yout, double *outmag,
                double *tx, double *ty, 
                double *im, double *gmag,
                int height, int width) {

    // calculate pixels' location
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    // Only execute valie pixels
    if (x>=width || y>=height) {
        return;
    }
    // get perpendicular line centered at current point
    double ctrp[2] = {y,x};
    // vector orthogonal to [ctrx,ctry]
    double vec[2] = {tx[y*width+x],ty[y*width+x]};
    double p_line[2][2*LEN+1] = {0.0};
    line2D(p_line, ctrp, vec);
    
    /*
    int idx_line[2][2*LEN+1] = {0};
    for (int i = 0; i < 2*LEN+1; i++){
        idx_line[0][i] = (int)(p_line[0][i]+0.5);
        idx_line[1][i] = (int)(p_line[1][i]+0.5);
    }
    */

    double sum_wd = 0.0;
    double temp[2] = {0.0};
    for (int i = 0; i < 2*LEN+1; i++){
        if (p_line[0][i]>=0 && p_line[0][i]<width && 
            p_line[1][i]>=0 && p_line[1][i]<height){
            int posx = (int)(p_line[0][i]+0.5);
            int posy = (int)(p_line[1][i]+0.5);
            int ind = posy*width+posx; 
            double ctrV[2] = {tx[y*width+x],ty[y*width+x]};
            double winV[2] = {tx[ind],ty[ind]};
            double wd = dire_weight(ctrV, winV); 
            sum_wd += wd;
            double wm = mag_weight(gmag[y*width+x], gmag[ind]);
            temp[0] += wd*wm*tx[ind];
            temp[1] += wd*wm*ty[ind];
        }
    }
    if (sum_wd/9<-0.1){
        temp[0] = -temp[0];
        temp[1] = -temp[1];
    }

    double temp_mag = sqrt(temp[0]*temp[0]+temp[1]*temp[1]);
    if (temp_mag != 0){
        outmag[y*width+x] = temp_mag;
        xout[y*width+x] = temp[0]/temp_mag;
        yout[y*width+x] = temp[1]/temp_mag;
    }
    __syncthreads(); 
}
